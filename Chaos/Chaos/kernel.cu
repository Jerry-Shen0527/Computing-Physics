
#include "hip/hip_runtime.h"

#include<cmath>
#include<iostream>
#include <stdio.h>
#include<iomanip>
#include<fstream>

using namespace std;

#define precision 1E-8
#define PI 3.141592653589793238462643383
#define divide (8192)
#define saving 8192

#define lambdamax 1.5
#define dlambda (double(lambdamax)/divide)

const int times = 1000000;

__global__ void kernel(double *dev_arr)//对每一个线程进行计算
{
	int offset = blockDim.x*blockIdx.x + threadIdx.x;//位置移动
	double thread_lambda = offset * dlambda;

	int start = offset * saving;

	for (int i = 0; i < times; i++)
	{
		dev_arr[start + (i) % saving] = thread_lambda * sin(PI * dev_arr[start + (i - 1) % saving]);
	}

}

double arr[divide*saving];//每一个线程会有一个大小为1024个数的buffer

int main()
{	

	ofstream out("C:\\Users\\10069\\Desktop\\Sinx.txt");
	double *dev_arr;

	hipMalloc((void**)&dev_arr, sizeof(double)*divide*saving);

	for (int i = 0; i < divide*saving; i++)
	{
		arr[i] = rand()-(RAND_MAX/2);
	}
		
	hipMemcpy(dev_arr,arr, sizeof(double)*divide*saving, hipMemcpyHostToDevice);
	

	int blocksize = 512;

	kernel<<<divide / blocksize, blocksize >>>(dev_arr);

	hipMemcpy(arr, dev_arr, sizeof(double)*divide*saving, hipMemcpyDeviceToHost);

	double result = 0;
	out << setprecision(12);
	for (int i = 0; i < divide*saving; i++)
	{
		out << arr[i] << ' ';
	}

	hipFree(dev_arr);
	out.close();

	system("pause");
}

